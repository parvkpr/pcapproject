#include "hip/hip_runtime.h"
__global__ void vecAdd(char* a, int width,int height, char* b)
{
	int row=blockIdx.x;
	int id=blockIdx.x*blockDim.x+threadIdx.x;
	int col=threadIdx.x;
if(row==0||row==width-1||col==width-1||col==0)
	{
		b[id]=a[id];
	}
else{
	int sum=0;
	int num=0;

				for(int j=-1;j<=1;j++)
				{
					for(int k=-1;k<=1;k++)
					{
						if (row+j >= 0 && row+j < width && col+k>= 0 && col+k< height)
						{sum=sum+a[(row+j)*height+col+k]; num++;}
					}
				}
				int q=sum/num;

				b[id]=q;

		}

}
#include<stdio.h>
#include<stdlib.h>
#include <stdint.h>
#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"
#define CHANNEL_NUM 3

int main(int *argc,char *argv[])
{
char* d_a;
char* d_o;
int width, height, bpp;
uint8_t *rgb_image = stbi_load("index.png", &width, &height, &bpp, 1);
/*for(int i=0;i<width;i++)
{
	for(int j=0;j<height;j++)
	{
		printf("%d \t",rgb_image[i*width+j]);
	}
	printf("\n");
}*/
//printf("%d %d %d: \n",rgb_image[0],width,height);
//stbi_image_free(rgb_image);
//int w = 800;
  //  int h = 800;

    //uint8_t* rb_image;
    //rb_image = malloc(width*height*CHANNEL_NUM);

    // Write your code to populate rgb_image here

    //stbi_write_png("image.png", width, height, CHANNEL_NUM, rgb_image, width*CHANNEL_NUM);
//h_a=(int *)malloc(sizeof(int));
//h_b=(int *)malloc(sizeof(int));
//h_c=(int *)malloc(n2*sizeof(int));

hipMalloc((void **)&d_a, width*height*sizeof(char));
//hipMalloc((void **)&d_b, sizeof(int));
//hipMalloc((void **)&d_c, sizeof(int));
hipMalloc((void **)&d_o, width*height*sizeof(char));
hipMemcpy( d_a,rgb_image, width*height*sizeof(char), hipMemcpyHostToDevice);
//hipMemcpy( d_b, &width, sizeof(int), hipMemcpyHostToDevice);
//hipMemcpy( d_c, &height,sizeof(int), hipMemcpyHostToDevice);

vecAdd<<<height,width>>>(d_a,width,height,d_o);
uint8_t *rb_image;
rb_image = (uint8_t *)malloc(width*height*sizeof(char));

hipMemcpy( rb_image, d_o,width*height*sizeof(char), hipMemcpyDeviceToHost);
printf("output ksjsksjsj\n \n \n \n");
for(int i=0;i<width;i++)
{
	for(int j=0;j<height;j++)
	{
		printf("%d \t",rb_image[i*width+j]);
	}
	printf("\n");
}
stbi_write_png("iaa.png", width, height, 1, rb_image, width*1);
stbi_write_png("ima.png", width, height, 1, rgb_image, width*1);
/*for(int i=0;i<n2;i++)
{
	if(i%col1==0&&i!=0)
			printf("\n");
	printf("%d ",h_c[i]);

}*/
}

